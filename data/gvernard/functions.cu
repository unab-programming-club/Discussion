#include "hip/hip_runtime.h"
#include "functions.hpp"


__global__ void kernelChi2(float* d_mua,float* d_mub,float* d_like_dum,int Nwave,int Nlocs);
__constant__ float const_f_obs[64];
__constant__ float const_df_obs[64];


void getChi2Cuda(double* like_all,float** f_obs_all,float* df_obs,float** h_mua,float** h_mub,int Nlocs,int Nwave,int Nratios){
  hipError_t err;

  // Size of the shared memory required in the kernel is always Nwave*Nthreads*sizeof(float)
  // i.e. this is the chunk of muA locations for all wavelengths.
  // I need to generally keep this below 48KB.
  int Ngrid;
  int Nthreads;
  setGridThreads(Ngrid,Nthreads,Nwave);
  printf("Shared memory occupancy: %d/48000\n",Nwave*Nthreads*sizeof(float));
  printf("Ngrid/Nthreads = %d/%d\n",Ngrid,Nthreads);


  // Allocate memory on device
  // inputs:
  float* d_mua;
  float* d_mub;
  hipMalloc(&d_mua,Nwave*Nlocs*sizeof(float));
  hipMalloc(&d_mub,Nwave*Nlocs*sizeof(float));

  // output:
  int Nlike = Ngrid*Nthreads;
  float* h_like_dum = (float*) malloc(Nlike*sizeof(float));
  float* d_like_dum;
  hipMalloc(&d_like_dum,Nlike*sizeof(float));
  err = hipGetLastError();
  if( err != hipSuccess ){
    fprintf(stderr,"Error: %s - in \"memory allocation\" \n",hipGetErrorString(err));
  }




  // Transfer (input) memory from host to device
  for(int k=0;k<Nwave;k++){
    hipMemcpy(d_mua+k*Nlocs,h_mua[k],Nlocs*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_mub+k*Nlocs,h_mub[k],Nlocs*sizeof(float),hipMemcpyHostToDevice);
  }
  hipMemcpyToSymbol(HIP_SYMBOL(const_df_obs),(void*)df_obs,Nwave*sizeof(float),0);
  err = hipGetLastError();
  if( err != hipSuccess ){
    fprintf(stderr,"Error: %s - in \"memory transfer to device\" \n",hipGetErrorString(err));
  }




  // Loop over f_obs (Nratios times)
  dim3 grid(Ngrid);
  dim3 threads(Nthreads);
  int Nmem = Nwave*Nthreads;
  for(int i=0;i<Nratios;i++){

    // Transfer f_obs to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(const_f_obs),(void*)f_obs_all[i],Nwave*sizeof(float),0);


    // Execute GPU kernels
    kernelChi2<<<grid,threads,Nmem*sizeof(float)>>>(d_mua,d_mub,d_like_dum,Nwave,Nlocs);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if( err != hipSuccess ){
      fprintf(stderr,"Error: %s - in \"kernelChi2\" \n",hipGetErrorString(err));
    }

    // Get memory from device
    hipMemcpy(h_like_dum,d_like_dum,Nlike*sizeof(float),hipMemcpyDeviceToHost);
    err = hipGetLastError();
    if( err != hipSuccess ){
      fprintf(stderr,"Error: %s - in \"memory transfer from device\" \n",hipGetErrorString(err));
    }


    // Add all the likelihoods in h_like_dum
    double like = 0.0;
    for(int j=0;j<Nlike;j++){
      like += h_like_dum[j];
    }
    like_all[i] = like;

    std::cout << like_all[i] << std::endl;
  }


  free(h_like_dum);
}




__global__ void kernelChi2(float* d_mua,float* d_mub,float* d_like_dum,int Nwave,int Nlocs){
  unsigned int t         = threadIdx.x;
  unsigned int Nthreads  = blockDim.x;
  unsigned int thread_id = blockIdx.x*Nthreads + t;
  unsigned int Nblocks   = gridDim.x;

  // Allocate shared memory
  extern __shared__ float mua[];


  float like = 0.0;
  // Loop over all of muA, reading it block by block
  for(int j=0;j<Nblocks;j++){

    // Each thread reads Nwave entries from muA into shared memory (different wavelengths from the same magmap location)
    for(int k=0;k<Nwave;k++){
      mua[k*Nthreads+t] = d_mua[k*Nlocs+Nthreads*j+t];
    }
    __syncthreads();

   
    // Each thread combines its unique muB value (thread_id) with the muA values currently into shared memory
    for(int i=0;i<Nthreads;i++){
      float fac = 0.0;
      for(int k=0;k<Nwave;k++){
	float fsim = mua[k*Nthreads+i]/d_mub[k*Nlocs+thread_id];
	float dum = (const_f_obs[k] - fsim)/const_df_obs[k];
	fac += dum*dum/2.0;
      }
      float dum = exp(-fac);
      like += dum;
      //      like += 1;
    }

  }
  
  d_like_dum[thread_id] = like;
}


void setGridThreads(int& Ngrid,int& Nthreads,int Nwave){
  Ngrid = 10;
  Nthreads = 1000;
  int shared_mem_size = Nthreads*Nwave*sizeof(float);

  while( shared_mem_size > 30000 ){
    Ngrid *= 2;
    Nthreads /= 2;
    shared_mem_size = Nthreads*Nwave*sizeof(float);
  }
}






void getChi2CudaCPU(double* like_all,float** f_obs_all,float* df_obs,float** h_mua,float** h_mub,int Nlocs,int Nwave,int Nratios){

  for(int q=0;q<Nratios;q++){

    double like = 0.0;
    for(int i=0;i<Nlocs;i++){
      for(int j=0;j<Nlocs;j++){

	double chi2 = 0.0;
	for(int k=0;k<Nwave;k++){
	  double fsim = h_mua[k][i]/h_mub[k][j];
	  double dum = (f_obs_all[q][k] - fsim)/df_obs[k];
	  chi2 += dum*dum;
	}

	like += exp(-chi2/2.0);
      }
    }

    like_all[q] = like;
    std::cout << like_all[q] << std::endl;
  }



}

